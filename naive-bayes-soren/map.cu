#include "hip/hip_runtime.h"
/*$Id: map.cu 2011-06-25 13:22:54$*/
/**
 *This is the source code for Soren, an Adaptive MapReduce framework on graphics
 *processors.
 *Developer: Reza Mokhtari(Shiraz University) 
 *If you have any question on the code, please contact us at 
 *           rmokhtari@cse.shirazu.ac.ir
 * */

/******************************************************************
 *Naive Bayes (NB) Developer: Saeed Kazemi <kazemi.ms@gmail.com>
 ******************************************************************/

#ifndef __MAP_CU__
#define __MAP_CU__

#include "../native_common/SorenInc.h"
#include "global.h"

__device__ int d_isdigit (char a)
{
	return '0' <= a && a <= '9';
}
__device__ long d_atoi(char* ptr, int size)
{
	long result = 0;
	for (int i = 0; i < size; i++)
		result = result * 10 + (ptr[i] - '0');
	return result;
}
__device__ int d_itoa(char* ptr, long val)
{
	int i;
	long pow = 1;
	for (i = 0; i < 19 && pow <= val; i++)
		pow *= 10;
	if (pow > val)
		pow /= 10;
	for (i = 0; pow > 0; i++)
	{
		ptr[i] = val / pow + '0';
		val = val % pow;
		pow /= 10;
	}
	if(!i)
		ptr[i++] = '0';
	ptr[i] = '\0';
	return i;
}
__device__ long calculateMapKey(int effectColumn, int effectValue, int causeValue)
{
	return causeValue * 100000 + effectColumn * 100 + effectValue;
}
__device__ int hash_func(char* str, int len)
{
	int hash, i;
	for (i = 0, hash=len; i < len; i++)
		hash = (hash<<4)^(hash>>28)^str[i];
	return hash;
}
__device__ void MAP_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	NB_VAL_T* nb_val = (NB_VAL_T*) val;
	if (nb_val->phase == 0)
	{
		int n, k;
		char* p = (char*)key;
		for (int i = 1; i <= nb_val->count; i++)
		{
			for(; !d_isdigit(*p); p++)
				;
			for(n = 0; d_isdigit(p[n]); n++)
				;
			if(i == nb_val->count)
				k = d_atoi(p, n);
			p += n;
		}
		p = (char*)key;
		int nk;
		char buffer[32];
		NB_VAL_T* o_val = (NB_VAL_T*)GET_OUTPUT_BUF(0);
		for (int i = 1; i <= nb_val->count - 1; i++)
		{
			for(; !d_isdigit(*p); p++)
				;
			for(n = 0; d_isdigit(p[n]); n++)
				;
			int j = d_atoi(p, n);
			p += n;
			o_val->count = 1;
			o_val->phase = 0;
			nk = d_itoa(buffer, calculateMapKey(i, j, k)) + 1;
			EMIT_INTERMEDIATE_FUNC(buffer, o_val, nk, sizeof(NB_VAL_T));
		}
		o_val->count = 1;
		o_val->phase = 0;
		nk = d_itoa(buffer, calculateMapKey(0, 0, k)) + 1;
		EMIT_INTERMEDIATE_FUNC(buffer, o_val, nk, sizeof(NB_VAL_T));
	}
	else 
	{
	}
}
#endif //__MAP_CU__
